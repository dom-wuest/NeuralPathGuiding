#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *//*
 */

/** @file   mlp-learning-an-image.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  Sample application that uses the tiny cuda nn framework to learn a
            2D function that represents an image.
 */

#include <tiny-cuda-nn/common_device.h>

#include <tiny-cuda-nn/config.h>

#include "tinyexr_wrapper.h"

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>


using namespace tcnn;
using precision_t = network_precision_t;



GPUMemory<float> load_image(const std::string& filename, int& width, int& height) {
	float* out; // width * height * RGBA
	load_exr(&out, &width, &height, filename.c_str());

	GPUMemory<float> result(width * height * 4);
	result.copy_from_host(out);
	free(out); // release memory of image data

	return result;
}

template <typename T>
void save_image(const T* image, int width, int height, int n_channels, int channel_stride, const std::string& filename) {
	std::vector<T> host_data(width * height * n_channels);
	CUDA_CHECK_THROW(hipMemcpy(host_data.data(), image, host_data.size()*sizeof(T), hipMemcpyDeviceToHost));

	std::vector<float> float_host_data(host_data.size());
	for (size_t i = 0; i < host_data.size(); ++i) {
		float_host_data[i] = (float)host_data[i];
	}

	save_exr(float_host_data.data(), width, height, n_channels, channel_stride, filename.c_str());
}

template <uint32_t stride>
__global__ void eval_image(uint32_t n_elements, hipTextureObject_t texture, float* __restrict__ xs_and_ys, float* __restrict__ result) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	uint32_t output_idx = i * stride;
	uint32_t input_idx = i * 2;

	float4 val = tex2D<float4>(texture, xs_and_ys[input_idx], xs_and_ys[input_idx+1]);
	result[output_idx + 0] = val.x;
	result[output_idx + 1] = val.y;
	result[output_idx + 2] = val.z;

	for (uint32_t i = 3; i < stride; ++i) {
		result[output_idx + i] = 1;
	}
}

int main(int argc, char* argv[]) {
	if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
		std::cout << "Turing Tensor Core operations must be compiled with CUDA 10.2 Toolkit or later." << std::endl;
		return -1;
	}

	hipDeviceProp_t props;

	hipError_t error = hipGetDeviceProperties(&props, 0);
	if (error != hipSuccess) {
		std::cout << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
		return -1;
	}

	if (!((props.major * 10 + props.minor) >= 75)) {
		std::cout << "Turing Tensor Core operations must be run on a machine with compute capability at least 75."
					<< std::endl;
		return -1;
	}

	if (argc < 2) {
		std::cout << "USAGE: " << argv[0] << " " << "path-to-image.exr [path-to-optional-config.json]" << std::endl;
		std::cout << "Sample EXR files are provided in 'data/images'." << std::endl;
		return 0;
	}

	try {
		json config = {
			{"loss", {
				{"otype", "RelativeL2"}
			}},
			{"optimizer", {
				{"otype", "Adam"},
				// {"otype", "Shampoo"},
				{"learning_rate", 1e-2},
				{"beta1", 0.9f},
				{"beta2", 0.99f},
				{"l2_reg", 0.0f},
				// The following parameters are only used when the optimizer is "Shampoo".
				{"beta3", 0.9f},
				{"beta_shampoo", 0.0f},
				{"identity", 0.0001f},
				{"cg_on_momentum", false},
				{"frobenius_normalization", true},
			}},
			{"encoding", {
				{"otype", "OneBlob"},
				{"n_bins", 32},
			}},
			{"network", {
				{"otype", "FullyFusedMLP"},
				// {"otype", "CutlassMLP"},
				// {"otype", "CutlassResNet"},
				{"n_neurons", 64},
				{"n_layers", 4},
				{"activation", "ReLU"},
				{"output_activation", "None"},
			}},
		};

		if (argc >= 3) {
			std::cout << "Loading custom json config '" << argv[2] << "'." << std::endl;
			std::ifstream f{argv[2]};
			config = json::parse(f, nullptr, true, /*skip_comments=*/true);
		}

		// First step: load an image that we'd like to learn
		int width, height;
		GPUMemory<float> image = load_image(argv[1], width, height);

		// Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = image.data();
		resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
		resDesc.res.pitch2D.width = width;
		resDesc.res.pitch2D.height = height;
		resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.normalizedCoords = true;
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.addressMode[1] = hipAddressModeClamp;
		texDesc.addressMode[2] = hipAddressModeClamp;

		hipResourceViewDesc viewDesc;
		memset(&viewDesc, 0, sizeof(viewDesc));
		viewDesc.format = hipResViewFormatFloat4;
		viewDesc.width = width;
		viewDesc.height = height;

		hipTextureObject_t texture;
		CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, &viewDesc));

		// Third step: sample a reference image to dump to disk. Visual comparison of this reference image and the learned
		//             function will be eventually possible.

		int sampling_width = width;
		int sampling_height = height;

		// Uncomment to fix the resolution of the training task independent of input image
		// int sampling_width = 1024;
		// int sampling_height = 1024;

		uint32_t n_coords = sampling_width * sampling_height;
		uint32_t n_coords_padded = (n_coords + 255) / 256 * 256;

		GPUMemory<float> sampled_image(n_coords * 3);
		GPUMemory<float> xs_and_ys(n_coords_padded * 2);

		std::vector<float> host_xs_and_ys(n_coords * 2);
		for (int y = 0; y < sampling_height; ++y) {
			for (int x = 0; x < sampling_width; ++x) {
				int idx = (y * sampling_width + x) * 2;
				host_xs_and_ys[idx+0] = (float)(x + 0.5) / (float)sampling_width;
				host_xs_and_ys[idx+1] = (float)(y + 0.5) / (float)sampling_height;
			}
		}

		xs_and_ys.copy_from_host(host_xs_and_ys.data());

		linear_kernel(eval_image<3>, 0, nullptr, n_coords, texture, xs_and_ys.data(), sampled_image.data());

		save_image(sampled_image.data(), sampling_width, sampling_height, 3, 3, "reference.exr");

		// Fourth step: train the model by sampling the above image and optimizing an error metric

		// Various constants for the network and optimization
		const uint32_t batch_size = 1 << 16;
		const uint32_t n_training_steps = argc >= 4 ? atoi(argv[3]) : 10000000;
		const uint32_t n_input_dims = 2; // 2-D image coordinate
		const uint32_t n_output_dims = 3; // RGB color

		hipStream_t inference_stream;
		CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
		hipStream_t training_stream = inference_stream;

		default_rng_t rng{1337};

		// Auxiliary matrices for training
		GPUMatrix<float> training_target(n_output_dims, batch_size);
		GPUMatrix<float> training_batch(n_input_dims, batch_size);

		// Auxiliary matrices for evaluation
		GPUMatrix<float> prediction(n_output_dims, n_coords_padded);
		GPUMatrix<float> inference_batch(xs_and_ys.data(), n_input_dims, n_coords_padded);

		json encoding_opts = config.value("encoding", json::object());
		json loss_opts = config.value("loss", json::object());
		json optimizer_opts = config.value("optimizer", json::object());
		json network_opts = config.value("network", json::object());

		std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
		std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
		std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);

		auto trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		float tmp_loss = 0;
		uint32_t tmp_loss_counter = 0;

		std::cout << "Beginning optimization with " << n_training_steps << " training steps." << std::endl;

		for (uint32_t i = 0; i < n_training_steps; ++i) {
			bool print_loss = i % 1000 == 0;
			bool visualize_learned_func = argc < 5 && i % 1000 == 0;

			// Compute reference values at random coordinates
			{
				generate_random_uniform<float>(training_stream, rng, batch_size * n_input_dims, training_batch.data());
				linear_kernel(eval_image<n_output_dims>, 0, training_stream, batch_size, texture, training_batch.data(), training_target.data());
			}

			// Training step
			float loss_value;
			{
				trainer->training_step(training_stream, training_batch, training_target, &loss_value);
			}
			tmp_loss += loss_value;
			++tmp_loss_counter;

			// Debug outputs
			{
				if (print_loss) {
					std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
					std::cout << "Step#" << i << ": " << "loss=" << tmp_loss/(float)tmp_loss_counter << " time=" << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;

					tmp_loss = 0;
					tmp_loss_counter = 0;
				}

				if (visualize_learned_func) {
					network->inference(inference_stream, inference_batch, prediction);
					save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, std::to_string(i) + ".exr");
				}

				// Don't count visualizing as part of timing
				// (assumes visualize_learned_pdf is only true when print_loss is true)
				if (print_loss) {
					begin = std::chrono::steady_clock::now();
				}
			}
		}

		// Dump final image if a name was specified
		if (argc >= 5) {
			network->inference(inference_stream, inference_batch, prediction);
			save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, argv[4]);
		}
	} catch (std::exception& e) {
		std::cout << "Uncaught exception: " << e.what() << std::endl;
	}

	return EXIT_SUCCESS;
}

